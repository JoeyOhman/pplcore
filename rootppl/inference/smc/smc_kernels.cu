
#include <hip/hip_runtime.h>

/*
 * File smc_kernels.cu contains definitions of kernels used by SMC. 
 */

 #ifdef __NVCC__

 #include <curand_kernel.h>
 #include "inference/smc/smc.cuh"
 #include "smc_kernels.cuh"

__global__ void initCurandStates(curandState* randStates, int numThreads, int seed) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= numThreads || i < 0) return;

    // Double check this seed, need only to be unique over one inference, as time should vary between inferences. 
    // curand_init(1234 + clock64(), seed * numThreads + i, 0, &particles->randStates[i]);
    curandState randStateLocal = randStates[i];
    curand_init(1234 + clock64(), seed * numThreads + i, 0, &randStateLocal);
    randStates[i] = randStateLocal;
}

__global__ void execFuncs(curandState* randStates, particles_t particles, const pplFunc_t* funcs, 
                            int numParticles, int numThreads, int numBblocks, void* arg) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // if(i >= numParticles || i < 0) return;
    if(i >= numThreads || i < 0) return;

    curandState randStateLocal = randStates[i];
    
    for(int j = i; j < numParticles; j += numThreads) {
        // funcs[particles.pcs[i]](&randStateLocal, particles, i, arg);
        int pc = particles.pcs[j];
        if(pc < numBblocks && pc >= 0)
            funcs[pc](&randStateLocal, particles, j, arg);
    }

    randStates[i] = randStateLocal;
}

#endif
