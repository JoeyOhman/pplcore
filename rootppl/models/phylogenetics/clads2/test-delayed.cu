#include "hip/hip_runtime.h"
/**
 * Tests waitingTimeDelayed
 */

#include <iostream>
#include <cstring>
#include <string>
#include <fstream>

#include "inference/smc/smc.cuh"
#include "utils/math.cuh"
#include "delayed.cuh"


const floating_t k = 1000;
const floating_t theta = 0.0001;
const floating_t factor = 2;
const floating_t observedTime = 10;
const floating_t nEvents = 5;
const floating_t elapsedTime = 10;


INIT_MODEL(floating_t, 1);



BBLOCK(testWaitingTime, {
  /* We will sample two waiting times (so that we have an update on the rate),
     and then check the distribution of the second waiting time against WebPPL.*/
  floating_t lambda = SAMPLE(gamma, k, theta);
  //printf("%f", lambda*factor);
  floating_t t0 = SAMPLE(exponential, lambda*factor);
  floating_t t1 = SAMPLE(exponential, lambda*factor);
       
  PSTATE = t0 + t1;
  PC++;
});




BBLOCK(testWaitingTimeDelayed, {
    /* We will sample two waiting times (so that we have an update on the rate),
       and then check the distribution of the second waiting time against WebPPL.*/
    rate_t lambda(k, theta);

    ret_delayed_t ret0 = BBLOCK_CALL(sampleWaitingTimeDelayed, lambda.k, lambda.theta, factor);
    ret_delayed_t ret1 = BBLOCK_CALL(sampleWaitingTimeDelayed, ret0.k, ret0.theta, factor);
        
    PSTATE = ret0.res + ret1.res;
    
    PC++;
  });


BBLOCK(testWaitingTimeDelayedRef, {
    /* We will sample two waiting times (so that we have an update on the rate),
       and then check the distribution of the second waiting time against WebPPL.*/
    rate_t lambda(k, theta);
    //ret_delayed_t ret0 = BBLOCK_CALL(waitingTimeDelayed, lambda.k, lambda.theta, lambda.factor);
    //ret_delayed_t ret1 = BBLOCK_CALL(waitingTimeDelayed, ret0.k, ret0.theta, lambda.factor);
   
    floating_t t0 = BBLOCK_CALL(sampleWaitingTimeDelayedRef, lambda, factor);
    floating_t t1 = BBLOCK_CALL(sampleWaitingTimeDelayedRef, lambda, factor);
    
    PSTATE = t0 + t1;
    
    PC++;
  });





  BBLOCK(testObserveWaitingTime, {
    floating_t lambda = SAMPLE(gamma, k, theta);
    OBSERVE(exponential, lambda*factor, observedTime);

    floating_t t0 = SAMPLE(exponential, lambda*factor);
    PSTATE = t0;
    PC++;
  });





BBLOCK(testObserveWaitingTimeDelayed, {
    rate_t lambda(k, theta);
    ret_delayed_t ret0 = BBLOCK_CALL(observeWaitingTimeDelayed, observedTime, lambda.k, lambda.theta, factor);
    WEIGHT(ret0.res);

    ret_delayed_t ret1 = BBLOCK_CALL(sampleWaitingTimeDelayed, ret0.k, ret0.theta, factor);
    PSTATE = ret1.res;
    PC++;
  });





BBLOCK(testObserveWaitingTimeDelayedRef, {
    rate_t lambda(k, theta);
    floating_t ret0 = BBLOCK_CALL(observeWaitingTimeDelayedRef, observedTime, lambda, factor);
    WEIGHT(ret0);

    floating_t ret1 = BBLOCK_CALL(sampleWaitingTimeDelayedRef, lambda, factor);
    PSTATE = ret1;
    PC++;
  });





BBLOCK(testObserveXEvents, {
    floating_t lambda = SAMPLE(gamma, k, theta);
    
    OBSERVE(poisson, lambda*factor*elapsedTime, nEvents);

    floating_t t0 = SAMPLE(exponential, lambda*factor);
    
    PSTATE = t0;
    PC++;
  });


BBLOCK(testObserveXEventsDelayed, {
  rate_t lambda(k, theta);

  ret_delayed_t ret0 = BBLOCK_CALL(observeXEventsDelayed, nEvents, elapsedTime, lambda.k, lambda.theta, factor);

  WEIGHT(ret0.res);

  ret_delayed_t ret1 = BBLOCK_CALL(sampleWaitingTimeDelayed, lambda.k, lambda.theta, factor);

  PSTATE = ret1.res;
  PC++;
  });


BBLOCK(testObserveXEventsDelayedRef, {
    rate_t lambda(k, theta);
    
    floating_t ret0 = BBLOCK_CALL(observeXEventsDelayedRef, nEvents, elapsedTime, lambda, factor);
    
    WEIGHT(ret0);
    
    floating_t ret1 = BBLOCK_CALL(sampleWaitingTimeDelayedRef, lambda, factor);
    
    PSTATE = ret1;
    PC++;
  });




CALLBACK(stats, {
    double sum = 0;
    
    for(int i = 0; i < N; i++) {
        sum += PSTATES[i];
    }
    double mean = sum / N;

    double sumSqDist = 0;
    for(int i = 0; i < N; i++) {
      sumSqDist += (PSTATES[i] - mean)*(PSTATES[i] - mean);    
    }
    double var = sumSqDist / N;
    
    printf("Sample mean: %f\n", mean);
    printf("Sample var: %f\n", var);
    printf("Sample stddev: %f\n", sqrt(var));
})

MAIN({
    //ADD_BBLOCK(testWaitingTime);
    //ADD_BBLOCK(testWaitingTimeDelayed);
    //ADD_BBLOCK(testWaitingTimeDelayedRef);
    
    //ADD_BBLOCK(testObserveWaitingTime);
    //ADD_BBLOCK(testObserveWaitingTimeDelayed);
    //ADD_BBLOCK(testObserveWaitingTimeDelayedRef);

    //ADD_BBLOCK(testObserveXEvents);
    //ADD_BBLOCK(testObserveXEventsDelayed);
    ADD_BBLOCK(testObserveXEventsDelayedRef);
    SMC(stats);
  })
