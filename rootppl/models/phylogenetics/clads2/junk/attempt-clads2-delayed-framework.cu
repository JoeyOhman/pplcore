#include "hip/hip_runtime.h"
/*
 * ClaDS2 Model
 * - uses the framework
 * - delayed sampling for lambda0
 *
 */

#include <iostream>
#include <cstring>
#include <string>
#include <fstream>

#include "inference/smc/smc.cuh"
#include "../tree-utils/tree_utils.cuh"
#include "utils/math.cuh"
#include "utils/stack.cuh"
#include "dists/delayed.cuh"

typedef bisse32_tree_t tree_t;
//typedef primate_tree_t tree_t;
//typedef moth_div_tree_t tree_t;


/**
 * The delayed program state now has the latest k
 * and theta, as well as a stack of factors;
 */
typedef short treeIdx_t;
struct progStateDelayed_t {
  pStack_t stack;
  gamma_t lambda_0;
  floating_t lambda0;
  normalInverseGamma_t alphaSigma;
  
  floating_t alpha;
  floating_t sigma;
  floating_t epsilon;
  floating_t rho;
  treeIdx_t treeIdx;
};

struct simBranchRet_t {
    floating_t r0;
    floating_t r1;
    floating_t r2;

    DEV simBranchRet_t(){};

    DEV simBranchRet_t(floating_t r0_, floating_t r1_, floating_t r2_) {
        r0 = r0_;
        r1 = r1_;
        r2 = r2_;
    }
};



/**
 * This function simulates the side-branches and returns 
 * true if the side branch does not make it to our sapmle.
 *
 * - start_time: when did the lineage speciate
 * - lambda0  (as reference to be updated)
 * - factor: the accummulated lambda factor at start_time
 * - alpha
 * - sigma
 * - epsilon
 * - rho
 * 
 * Returns: TRUE/FALSE
 * also it has side effect that the proposal for lambda0 is updated
 */
BBLOCK_HELPER(clads2GoesUndetectedDelayed, {
    
    // extreme values patch
    if (factor > 1e5) {
      return false; // detected for sure with insane div. rate
    }
    
    if (factor < 1e-5) {
      // lambda is very small, so nothing will happen to the lineage in terms of speciation
      // it will hit present and then we see
      bool undetected = !SAMPLE(bernoulli, rho);  
      return undetected;
    }
    // end extreme values patch 1
    
    // t is the waiting time until the next event (speciation or extinction)
    // TODO is the next line correct?
    floating_t t = sample_GammaExponential(lambda0, factor*(1 + epsilon));
    
    floating_t currentTime = startTime - t;
    
    if(currentTime < 0) { // we are in the future, rho is the detection probability
        bool undetected = !SAMPLE(bernoulli, rho);  
        return undetected;
    }
            
    //  bool exctinction = SAMPLE(bernoulli, mu / (mu + lambda));
    bool speciation = SAMPLE(bernoulli, 1.0/(1.0 + epsilon));
    bool extinction = !speciation;
    
    if(extinction) {
      return true;
    }
    
    // Realizes the new factor by which the current lambda (= lambda_0 x old factors)
    // is going to be multiplied. One for left and right.
    floating_t f1 = sample_NormalInverseGammaNormal(alphaSigma);
      //SAMPLE(normal, log(alpha), sigma);
    floating_t f2 = sample_NormalInverseGammaNormal(alphaSigma);
      //SAMPLE(normal, log(alpha), sigma);

    //rate_t leftRate(lambdaRate.k, lambdaRate.theta, lambdaRate.factor*exp(f1));

    bool ret1 = BBLOCK_CALL(clads2GoesUndetectedDelayed, currentTime, lambda0, factor*exp(f1), alphaSigma, epsilon, rho);
    
    bool leftDetection = !ret1;
    if (leftDetection) return ret1; // no need to descend to the right side of the tree
    
    //rate_t rightRate(ret1.rate_upd.k, ret1.rate_upd.theta, lambdaRate.factor*exp(f2));

    bool ret2 = BBLOCK_CALL(clads2GoesUndetectedDelayed, currentTime, lambda0, factor*exp(f2), alphaSigma, epsilon, rho);
    return ret2;
    
  }, bool, floating_t startTime, gamma_t& lambda0, floating_t factor, normalInverseGamma_t& alphaSigma, floating_t epsilon, floating_t rho)




// Simulates the hidden speciation events along a branch
// Returns
//   - the accumulated probability along the branch
//   - the accumulated factors along the branch
// has side-effect that the proposal for lambda0 is updated
BBLOCK_HELPER(simBranchDelayed, {

    floating_t t1 = startTime - stopTime;
    
    // extreme values patch 2/2
    if (factor > 1e5 ) {
      simBranchRet_t ret(0.0, 0.0, -INFINITY);
      //printf("large factor");
    }

    if (factor < 1e-5) {
      floating_t ret0 = score_GammaPoisson(0, t1, lambda0, factor*(epsilon));
      simBranchRet_t ret(factor, 0.0, ret0);
    }
    // end extreme values patch 2/2

    floating_t t = sample_GammaExponential(lambda0, factor);
    floating_t currentTime = startTime - t;
    

    if(currentTime <= stopTime) {
      floating_t ret1 = score_GammaPoisson(0, t1, lambda0, factor*(epsilon));
      
      //floating_t ret2 = BBLOCK_CALL(observeWaitingTimeDelayedRef, 0, lambda0, factor);
      simBranchRet_t ret(factor, 0.0, ret1);
      return ret;
    }
    
    // sample factors for left and right subtrees
    floating_t f1 = sample_NormalInverseGammaNormal(alphaSigma);
      //SAMPLE(normal, log(alpha), sigma); // left factor
    floating_t f2 = sample_NormalInverseGammaNormal(alphaSigma);
      //SAMPLE(normal, log(alpha), sigma); // right factor
    
    // we need to check if the side was undetected
    // w.l.o.g. we choose the right side to die
    //rate_t rightRate(lambdaRate.k, lambdaRate.theta, lambdaRate.factor*exp(f2));
    bool sideUndetected = BBLOCK_CALL(clads2GoesUndetectedDelayed, currentTime, lambda0, factor*exp(f2), alphaSigma, epsilon, rho);

    if(! sideUndetected) {
      // this particle needs to die
      
      simBranchRet_t ret(0.0, 0.0, -INFINITY);
      return ret;
    }
    
    
    // Now we will enter into the recursion to process the rest of the branch
    // and accummulate the factor
    //    rate_t leftRate(lambdaRate.k, lambdaRate.theta, lambdaRate.factor*exp(f1));
    
    simBranchRet_t ret7 = BBLOCK_CALL(simBranchDelayed, currentTime, stopTime, lambda0, factor*exp(f1), alphaSigma, epsilon, rho);

    floating_t extinctionProb = score_GammaPoisson(0, t, lambda0, factor*(epsilon));  // branch didn't go extinct

    
    // Now gather all weights and add 2 for the end of the branch
    // 1 and 2 are probs, 3 is a bool, 4 is a prob again
    //simBranchRetDelayed_t rt(ret7.prob + extinctionProb + log(2.0), ret7.factor);
  simBranchRet_t rt(ret7.r0, 0.0, ret7.r2 + log(2.0) + extinctionProb);

  return rt;
    
  }, simBranchRet_t, floating_t startTime, floating_t stopTime, gamma_t& lambda0, floating_t factor, normalInverseGamma_t& alphaSigma, floating_t epsilon, floating_t rho);



 

#define NUM_BBLOCKS 4
INIT_MODEL(progStateDelayed_t, NUM_BBLOCKS)

BBLOCK_DATA(tree, tree_t, 1);

// Not called on root as in WebPPL, instead root is handled in simClaDS2 bblock
BBLOCK(simTree, {

    // Fetch tree data
    tree_t* treeP = DATA_POINTER(tree);
    treeIdx_t treeIdx = PSTATE.treeIdx;
    
    // Terminate if tree is fully traversed
    if(treeIdx == -1) {
      PC++;
      BBLOCK_CALL(DATA_POINTER(bblocksArr)[PC], NULL);
      return;
    }
    
    PSTATE.treeIdx = treeP->idxNext[treeIdx];
    
    int indexParent = treeP->idxParent[treeIdx];
    
    floating_t factor = PSTATE.stack.pop();
    
    simBranchRet_t ret = BBLOCK_CALL(simBranchDelayed, treeP->ages[indexParent], treeP->ages[treeIdx],  PSTATE.lambda_0, factor, PSTATE.alphaSigma, PSTATE.epsilon, PSTATE.rho);

    floating_t factorEnd = ret.r0;

    bool interiorNode = treeP->idxLeft[treeIdx] != -1 || treeP->idxRight[treeIdx] != -1;
    floating_t lnTerminalProb = interiorNode ? score_GammaExponential(0, PSTATE.lambda_0, factorEnd) : log(PSTATE.rho);
    //printf("%f %f\n", ret.r2, lnTerminalProb);
    WEIGHT(ret.r2 + lnTerminalProb);

    // Collect node and branch info, todo?
    
    if(interiorNode) {
      //floating_t lambda1 = LAMBDA_CHOOSER(lambda, lambdaEnd, PSTATE.alpha, PSTATE.sigma);
      //floating_t lambda2 = LAMBDA_CHOOSER(lambda, lambdaEnd, PSTATE.alpha, PSTATE.sigma);
      floating_t f1 = sample_NormalInverseGammaNormal(PSTATE.alphaSigma);
      printf("%f %f\n", f1, exp(f1));
	//SAMPLE(normal, log(PSTATE.alpha), PSTATE.sigma);
      floating_t f2 = sample_NormalInverseGammaNormal(PSTATE.alphaSigma);
	//SAMPLE(normal, log(PSTATE.alpha), PSTATE.sigma);
      floating_t leftf = factorEnd*exp(f1);
      floating_t rightf = factorEnd*exp(f2);
      
      PSTATE.stack.push(rightf);
      PSTATE.stack.push(leftf);
    }
    
})
 
 
BBLOCK(simClaDS2, {
    tree_t* treeP = DATA_POINTER(tree);

    // Make sure this is the correct starting point
    PSTATE.treeIdx = treeP->idxLeft[ROOT_IDX];
 
    // Test settings
    /*
    floating_t lambda_0 = 0.2;
    floating_t alpha    = 1.0;
    floating_t sigma    = 0.0000001;
    floating_t epsilon  = 0.5;   // Corresponds to mu = epsilon*lambda = 0.1
    */
    floating_t rho      = 1.0;

    floating_t k = 1;
    floating_t theta = 0.2;

    floating_t m0 = 0;
    floating_t v = 1;
    floating_t a = 1.0;
    floating_t b = 0.2;
    

   // floating_t lambda_0 = SAMPLE(gamma, k, theta);
    gamma_t lambda_0(k, theta);

    //floating_t sigmaSquared = 1.0 / SAMPLE(gamma, 1.0, 1.0 / 0.2);
    //floating_t sigma = sqrt(sigmaSquared);
    //floating_t alpha = exp(SAMPLE(normal, 0.0, sigma));
    
// * σ^2 | a,b ~ InverseGamma(a, b)
// * m ~ N(m0, v σ^2)
    normalInverseGamma_t alphaSigma(m0, v, a, b);
    
    floating_t epsilon = SAMPLE(uniform, 0.0, 1.0);
    floating_t factor = 1.0;
    
    PSTATE.lambda_0 = lambda_0;
    //PSTATE.alpha = alpha;
    //PSTATE.sigma = sigma;
    PSTATE.alphaSigma = alphaSigma;
    PSTATE.epsilon = epsilon;
    PSTATE.rho = rho;
 
    // Correction Factor
    int numLeaves = countLeaves(treeP->idxLeft, treeP->idxRight, treeP->NUM_NODES);
    floating_t corrFactor = (numLeaves - 1) * log(2.0) - lnFactorial(numLeaves);
    //printf("correctionFactor: %f\n", corrFactor);
    WEIGHT(corrFactor);

    //floating_t lambda1 = LAMBDA_CHOOSER(lambda_0, lambda_0, alpha, sigma);
    //floating_t lambda2 = LAMBDA_CHOOSER(lambda_0, lambda_0, alpha, sigma);
    // TODO the following with a sample MACRO
    floating_t f1 = sample_NormalInverseGammaNormal(PSTATE.alphaSigma);
    // printf("%f %f\n", f1, exp(f1));
      //SAMPLE(normal, log(alpha), sigma);
    floating_t f2 = sample_NormalInverseGammaNormal(PSTATE.alphaSigma);
      // SAMPLE(normal, log(alpha), sigma);
    floating_t leftf = factor*exp(f1);
      //factor*exp(f1);
    floating_t rightf = factor*exp(f2);
      //factor*exp(f2);

    // bblockArgs_t args(lambda1, lambda2);
    // one for each child, and one for Survivorship Bias after tree simulations
    PSTATE.stack.push(rightf);
    PSTATE.stack.push(leftf);
    PSTATE.stack.push(factor);

    PC++;
    BBLOCK_CALL(simTree);

    // Condition on detection (clads2GoesUndetected simulations)
    // Nested inference with "forward" method here, just simulation with WEIGHT( -2.0 * log(number of false))?
})


// Should be equivalent to forward sampling
BBLOCK(conditionOnDetection, {

    tree_t* treeP = DATA_POINTER(tree);
    floating_t treeAge = treeP->ages[ROOT_IDX];

    gamma_t lambda0 = PSTATE.lambda_0;
    floating_t factor = PSTATE.stack.pop();

    //floating_t alpha = PSTATE.alpha;
    //floating_t sigma = PSTATE.sigma;
    normalInverseGamma_t alphaSigma = PSTATE.alphaSigma;
    floating_t epsilon = PSTATE.epsilon;
    floating_t rho = PSTATE.rho;

    int numSamples = 100;
    int numDetected = 0;
    for(int i = 0; i < numSamples; i++) {
        bool undetected = BBLOCK_CALL(clads2GoesUndetectedDelayed, treeAge, lambda0, factor, alphaSigma, epsilon, rho);
        if(! undetected)
            numDetected++;
    }
    //printf("condition weihght: %f", -2.0 * log(numDetected / static_cast<floating_t>(numSamples)) );
    WEIGHT(-2.0 * log(numDetected / static_cast<floating_t>(numSamples)));

    PC++;

})


BBLOCK(sampleFinalLambda, {
    PSTATE.lambda0 = SAMPLE(gamma, PSTATE.lambda_0.k, PSTATE.lambda_0.theta);
    floating_t sigmaSquared = 1.0 / SAMPLE(gamma, PSTATE.alphaSigma.a, 1.0 / PSTATE.alphaSigma.b);
    PSTATE.sigma = sqrt(sigmaSquared);
    PSTATE.alpha = exp(SAMPLE(normal, PSTATE.alphaSigma.m0, PSTATE.alphaSigma.v * PSTATE.sigma));
    PC++;
})
 
// Write particle data to file
CALLBACK(saveResults, {
      std::string fileName = "results/clads2-delayed.csv";
      std::ofstream resultFile (fileName);
      resultFile << "lambda0 k theta sigma alpha epsilon weight\n";
      if(resultFile.is_open()) {

          for(int i = 0; i < N; i++)
              resultFile << 
                  PSTATES[i].lambda0 << " " << PSTATES[i].lambda_0.k << " " << PSTATES[i].lambda_0.theta << " " << PSTATES[i].sigma << " " << PSTATES[i].alpha << " " << PSTATES[i].epsilon << " " << 
                  exp(WEIGHTS[i]) << "\n";

          resultFile.close();
      } else {
          printf("Could not open file %s\n", fileName.c_str());
      }

  })

MAIN({

    ADD_BBLOCK(simClaDS2);
    ADD_BBLOCK(simTree);
    ADD_BBLOCK(conditionOnDetection);
    ADD_BBLOCK(sampleFinalLambda);
    SMC(saveResults);
    //SMC(NULL)
})
 
 
